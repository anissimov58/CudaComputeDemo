
#include "hip/hip_runtime.h"

#include <stdio.h>
#include <iostream>
#include <chrono>
#define cyclelength 100
#define checksize 10000000.0

using namespace std;

int CPU(int N)
{
	float* vectA = new float[N];
	float* vectB = new float[N];
	float* vectC = new float[N];
	float* vectY = new float[N];

	float checksumm = 0;
	//��������� ������� �������
	for (int i = 0; i < N; i++) {
		vectA[i] = static_cast<float>(i * 1.1);
		vectB[i] = static_cast<float>(i * 2.7) + 1;
		vectC[i] = static_cast<float>(i * 3.14);
	}

	auto startc = std::chrono::high_resolution_clock::now();

	//����������
	for (int j = 0; j < cyclelength; j++)
		for (int i = 0; i < N; i++)
			vectY[i] = ((vectA[i] * vectB[i]) + ((vectA[i] * vectA[i] * vectA[i]) * vectC[i] / vectB[i]));

	auto finishc = std::chrono::high_resolution_clock::now();
	auto duration = std::chrono::duration_cast<std::chrono::microseconds>(finishc - startc).count();
	cout << "Chrono: N = " << N << "; CPU Time = " << duration / 1000.0 << " ms\n";

	for (int i = 0; i < N; i++) {
		checksumm += vectY[i] / checksize;
		//cout << "Result[" << i << "]:=" << vectY[i] << endl;
	}

	printf("checksumm = %f\n\n\n", checksumm);

	delete[] vectA;
	delete[] vectB;
	delete[] vectC;
	delete[] vectY;
	return 0;
}

__global__ void addKernel(float* vectA, float* vectB, float* vectC, float* vectY, int S)
{
	int k;
	//����� ����� � ������ * ������ ����� + ����� ������ � ������� �����
	int idx_thread = blockIdx.x * blockDim.x + threadIdx.x;
	int m = 0;
	for (int j = 0; j < cyclelength; j++)
		for (k = 0; k < S; k++)
		{
			m = idx_thread * S + k;
			vectY[m] = ((vectA[m] * vectB[m]) + ((vectA[m] * vectA[m] * vectA[m]) * vectC[m] / vectB[m]));
		}

}

__global__ void fun_kernel(float* a, int s) {
	
}

int GPU(int N, int blocks, int blocksize)
{
	float* vectA = new float[N];
	float* vectB = new float[N];
	float* vectC = new float[N];
	float* vectY = new float[N];


	float* devA;
	float* devB; //��������� �� ������ ��� ���
	float* devC;
	float* devY; //��������� �� ������ ��� ���

	float elapsedTime; //��� �������� ������� ���������� CUDA

	//float checksumm = 0;

	hipEvent_t start, stop; //�������������� �������
	hipEventCreate(&start); //������������� ������� start
	hipEventCreate(&stop); //������������� ������� stop

	float m = 3;
	float checksumm = 0;
	//��������� ������� �������
	for (int i = 0; i < N; i++) {
		vectA[i] = static_cast<float>(i * 1.1);
		vectB[i] = static_cast<float>(i * 2.7) + 1;
		vectC[i] = static_cast<float>(i * 3.14);
	}
	////��������� ������ �� ��� ��� ��������
	hipMalloc((void**)&devA, N * sizeof(float));
	hipMalloc((void**)&devB, N * sizeof(float));
	hipMalloc((void**)&devC, N * sizeof(float));
	hipMalloc((void**)&devY, N * sizeof(float));

	//blocks = 16; //���������� ������ �������
	//blocksize = 512; //������� �� ���� ����

	int steps = static_cast<int>(N / (blocks * blocksize));  //���������� ��������� ��������, ������� ������������ ������ �����

	//��������� ����� ������ ����������
	hipEventRecord(start, 0);
	auto startc = std::chrono::high_resolution_clock::now();

	//�������� ������ 1 �� ����� devA, �������� �����-��, � � ������ ��� �������� � ����� -> ���
	hipMemcpy(devA, vectA, N * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(devB, vectB, N * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(devC, vectC, N * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(devY, vectY, N * sizeof(float), hipMemcpyHostToDevice);

	//��������� �� ���
	addKernel <<<blocks, blocksize>>> (devA, devB, devC, devY, steps);

	//�������� ���� ����� ��������� ��������
	hipMemcpy(vectY, devY, N * sizeof(float), hipMemcpyDeviceToHost);

	hipFree(devA);
	hipFree(devB);
	hipFree(devC);
	hipFree(devY);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop); //������������� host � device �� ������� stop

	hipEventElapsedTime(&elapsedTime, start, stop); //����� = stop - start
	auto finishc = std::chrono::high_resolution_clock::now();
	auto duration = std::chrono::duration_cast<std::chrono::microseconds>(finishc - startc).count();
	cout << "Chrono: N = " << N << ";"<<" Blocks = "<<blocks<<"; Blocksize = "<<blocksize<<"; GPU Time = " << duration / 1000.0 << " ms\n";
	//cout << duration / 1000.0 << endl;
	printf("CUDA: GPU Time = %f\n", elapsedTime);

	for (int i = 0; i < N; i++) {
		checksumm += vectY[i] / checksize;
		//cout << "Result[" << i << "]:=" << vectY[i] << endl;
	}

	//printf("checksumm = %f\n\n\n", checksumm);

	//printf("vect1 = %5.f\n", vect1[1023]);

	//tracing tools such as Nsight and Visual Profiler to show complete traces.
	hipDeviceReset();

	delete[] vectA;
	delete[] vectB;
	delete[] vectC;
	delete[] vectY;

	return 0;
}

int main(){
	int count[3] = { pow(2,14),pow(2,16) };
	int blocks[5] = {1,2,4,8,16};
	int blocksize[6] = {1,4,32,64,256,512};

	for (int i = 0; i < 2; i++)
		{
			CPU(count[i]);
		}
	system("Pause");
	for (int i = 0; i < 2; i++) 
		for (int j = 0; j < 5; j++)
			for (int k = 0; k < 6; k++) 
				GPU(count[i], blocks[j], blocksize[k]);

	system("Pause");

	return 0;
}